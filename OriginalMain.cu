/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"
 
using namespace std;


ofstream outfile; // The handle for printing the output

/******************************Write your kerenels here ************************************/


       

    
    
    
    
/**************************************END*************************************************/



//Function to write result in output file
void printResult(int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ; // Number of vertices in the graph
    int E; // Number of edges in the graph
    int L; // number of levels in the graph

    //Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    //Parsing the graph to create csr list
    g.parseGraph();

    //Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();


    //Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    // Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    int *d_aid; // acive in-degree array
    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1)*sizeof(int));
    hipMalloc(&d_csrList, E*sizeof(int)); 
    hipMalloc(&d_apr, V*sizeof(int)); 
    hipMalloc(&d_aid, V*sizeof(int));

    //copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V*sizeof(int), hipMemcpyHostToDevice);

    // variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex;
    h_activeVertex = (int*)malloc(L*sizeof(int));
    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    // int *d_activeVertex;
	// hipMalloc(&d_activeVertex, L*sizeof(int));


/***Important***/

// Initialize d_aid array to zero for each vertex
// Make sure to use comments

/***END***/
double starttime = rtclock(); 

/*********************************CODE AREA*****************************************/







    
 

    
   
    
    

     

/********************************END OF CODE AREA**********************************/
double endtime = rtclock();  
printtime("GPU Kernel time: ", starttime, endtime);  

// --> Copy C from Device to Host
char outFIle[30] = "./output.txt" ;
printResult(h_activeVertex, L, outFIle);
if(argc>2)
{
    for(int i=0; i<L; i++)
    {
        printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
    }
}

    return 0;
}
