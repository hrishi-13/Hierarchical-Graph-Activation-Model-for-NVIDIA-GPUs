#include "hip/hip_runtime.h"
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */

 // all levels are calculated correctly

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"
 
using namespace std;
#define BLOCK_SIZE 1024

ofstream outfile; // The handle for printing the output

/******************************Write your kerenels here ************************************/

// initializing the vertexLevel array

__global__ void initializingVertexLevel(unsigned int* d_aid, int *d_apr, int *d_vertexLevel, unsigned int* d_activeVertex, int V, int L){

    __shared__ int s_vertexLevel[BLOCK_SIZE];

    int id = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(id >= 0 and id < V) {

        if(d_apr[id] > 0){
            s_vertexLevel[threadIdx.x] = L;
        }

        else if (d_apr[id] == 0){
            s_vertexLevel[threadIdx.x] = 0;
        }

        __syncthreads();
        d_vertexLevel[id] = s_vertexLevel[threadIdx.x];
    }
}

// finding which vertices are active and calculating the levels of each vertex

__global__ void calculatingVertexLevels(unsigned int *d_aid, int *d_apr, int *d_csrList, int *d_offset, int *d_vertexLevel, int V, int currentLevel) {

    int id = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(id >= 0 and id < V){

        if(d_vertexLevel[id] == currentLevel){
            int start_index = d_offset[id];
            int end_index = d_offset[id+1];

            int destinationLevel = currentLevel + 1;

            for(int i = start_index; i < end_index; i++) {
                int destination = d_csrList[i];
                d_vertexLevel[destination] = destinationLevel;
            } 

            // checking deactivation condition
            if(id >= 1 and id <= V-2){
                int prev = id - 1;
                int next = id + 1;
                if(d_aid[prev] < d_apr[prev] and d_aid[next] < d_apr[next] and d_vertexLevel[prev] == d_vertexLevel[next] ) {
                    d_aid[id] = 0;
                }
            }

            for(int i = start_index; i < end_index; i++) {
                // checking activation condition
                if(d_aid[id] >= d_apr[id]) {
                    int destination = d_csrList[i];
                    // increasing aid of vertex by 1
                    atomicAdd(&d_aid[destination], 1);
                }
            }
        }
    }
}

// counting no. of active vertices at each level

__global__ void countingActiveVertices(unsigned int *d_aid, int *d_apr, int *d_vertexLevel, unsigned int *d_activeVertex, int V, int noOfLevel) {

    unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(id >=0 and id < V){
        // active vertex condition
        if((d_aid[id] >= (unsigned) d_apr[id])){ 
            atomicInc(&d_activeVertex[d_vertexLevel[id]], V+1); 
        }
    }
}
    
/**************************************END*************************************************/

// Function to write result in output file

void printResult( int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ;   // Number of vertices in the graph
    int E;   // Number of edges in the graph
    int L;  // number of levels in the graph

    // Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    // Parsing the graph to create csr list
    g.parseGraph();

    // Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();

    // Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    //Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    unsigned int *d_aid; // acive in-degree array

    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1) * sizeof(int));
    hipMalloc(&d_csrList, E * sizeof(int)); 
    hipMalloc(&d_apr, V * sizeof(int)); 
    hipMalloc(&d_aid, V * sizeof(int));

    // copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V * sizeof(int), hipMemcpyHostToDevice);

    // variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex;
    h_activeVertex = (int*)malloc(L*sizeof(int));

    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    // int *d_activeVertex;
	// hipMalloc(&d_activeVertex, L*sizeof(int));


/***Important***/

// Initialize d_aid array to zero for each vertex
// Make sure to use comments

/***END***/
double starttime = rtclock(); 

/*********************************CODE AREA*****************************************/

// h_activeVertex already initialized
unsigned int *d_activeVertex;
hipMalloc(&d_activeVertex, L * sizeof(int));

// initializing active vertex array with 0
hipMemset(d_activeVertex, 0, L * sizeof(int));

int *d_vertexLevel; // contains levels for each vertex
hipMalloc(&d_vertexLevel, V * sizeof(int));

// initializing active in degree array with 0
hipMemset(d_aid, 0, V * sizeof(int));  

dim3 dimGrid1((V / BLOCK_SIZE) + 1, 1, 1); 
dim3 dimBlock1(BLOCK_SIZE, 1, 1);

// launching the kerneL: initializing the vertex level array
initializingVertexLevel<<<dimGrid1, dimBlock1>>>(d_aid, d_apr, d_vertexLevel, d_activeVertex, V, L);
hipDeviceSynchronize();

dim3 dimGrid2((V / BLOCK_SIZE ) + 1, 1, 1); 
dim3 dimBlock2(BLOCK_SIZE, 1, 1);

for(int currentLevel = 0; currentLevel < L; currentLevel++) {
    // launching the kerneL: calculating the vertex levels
    calculatingVertexLevels<<<dimGrid2, dimBlock2>>>(d_aid, d_apr, d_csrList, d_offset, d_vertexLevel, V, currentLevel);
}
hipDeviceSynchronize();

// launching the kernel: counting the number of active vertices at each level
countingActiveVertices<<<dimGrid2, dimBlock2>>>(d_aid, d_apr, d_vertexLevel, d_activeVertex, V, L);
hipDeviceSynchronize();

hipMemcpy(h_activeVertex, d_activeVertex, L * sizeof(int), hipMemcpyDeviceToHost);

/********************************END OF CODE AREA**********************************/

double endtime = rtclock();  
printtime("GPU Kernel time: ", starttime, endtime);  

// --> Copy C from Device to Host
char outFIle[30] = "./output.txt" ;
printResult(h_activeVertex, L, outFIle);

if(argc > 2){
    for(int i=0; i<L; i++){
        printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
    }
}

    return 0;
}